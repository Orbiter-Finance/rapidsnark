#include "groth16_cuda.cu"


int main(int argc, char **argv){
    std::cout << "Hello ICICLE" <<"\n";

    std::cout << "Hello CMAKE" << std::endl;
    u_int32_t log_ntt_size = 2;
    u_int32_t domainSize = 1 << log_ntt_size;
    FFT<AltBn128::Engine::Fr> fft(domainSize);
    auto a = new AltBn128::Engine::FrElement[domainSize];
    #pragma omp parallel for
    for (u_int32_t i = 0; i < domainSize; i++){
        Fr.fromUI(a[i], i + 1);
    }
    // auto input = reinterpret_cast<E *>(a);

    auto ctx = device_context::get_default_device_context();
    init_icicle_cuda_ntt_ctx(ctx, log_ntt_size);

    // const S basic_root = S::omega(log_ntt_size);
    // InitDomain(basic_root, ctx);

    NTTConfig<ICICLE_S> config = DefaultNTTConfig<ICICLE_S>();
    config.ntt_algorithm = NttAlgorithm::MixedRadix;
    // config.ntt_algorithm = NttAlgorithm::Radix2;
    config.batch_size = 1;

    START_TIMER(MixedRadix);
    // hipError_t err = NTT<S, E>(input, domainSize, NTTDir::kForward, config, input);
    hipError_t err = icicle_cuda_ntt(
        a,
        domainSize,
        config,
        NTTDir::kForward,
        ctx
    );

    END_TIMER(MixedRadix, "MixedRadix NTT");

    fft.printVector(&a[0], domainSize);
    delete [] a;

    return 0;
}