#include "hip/hip_runtime.h"
#ifndef __GROTH16_CUDA_CU__
#define __GROTH16_CUDA_CU__
#include <iostream>
#include <chrono>
#include <fstream>
#include <gmp.h>
#include <memory>
#include <stdexcept>


#include <nlohmann/json.hpp>

#include <alt_bn128.hpp>
#include "binfile_utils.hpp"
#include "zkey_utils.hpp"
#include "wtns_utils.hpp"
#include "groth16.hpp"

using json = nlohmann::json;

#define G2_DEFINED
#define CURVE_ID 1
#include "appUtils/ntt/ntt.cu"
#include "appUtils/ntt/kernel_ntt.cu"
#include "appUtils/msm/msm.cu"
using namespace curve_config;

// using namespace curve_config;
using namespace ntt;
using namespace AltBn128;
using namespace Groth16;

// Operate on scalars
typedef scalar_t ICICLE_S;
typedef scalar_t ICICLE_E;
using FpMilliseconds = std::chrono::duration<float, std::chrono::milliseconds::period>;
#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg) printf("%s: %.0f ms\n", msg, FpMilliseconds(std::chrono::high_resolution_clock::now() - timer##_start).count());

static void init_icicle_cuda_ntt_ctx(
    device_context::DeviceContext& ntt_ctx,
    const u_int32_t log_ntt_size
){
    const ICICLE_S basic_root = ICICLE_S::omega(log_ntt_size);
    InitDomain(basic_root, ntt_ctx);
}

// need to call init_ntt_ctx once!
static hipError_t icicle_cuda_ntt(
    Engine::FrElement* inoutput,
    u_int32_t ntt_size,
    NTTConfig<ICICLE_S> &config,
    NTTDir direction, // NTTDir::kForward for ntt, NTTDir::kInverse for intt
    device_context::DeviceContext ntt_ctx // 
){  
    auto rep_inoutput = reinterpret_cast<ICICLE_E *>(inoutput);

    return NTT<ICICLE_S, ICICLE_E>(
        rep_inoutput, 
        ntt_size, 
        direction, 
        config, 
        rep_inoutput
    );
}

static void bn254_icicle_cuda_g1_msm(
    Engine::G1Point &r,
    Engine::G1PointAffine *bases,
    uint8_t *scalars,
    unsigned int msm_size,
    msm::MSMConfig &config
){
    auto rep_bases = reinterpret_cast<affine_t *>(bases);
    auto rep_scalars = reinterpret_cast<scalar_t *>(scalars);
    projective_t temp;
    
    hipError_t res = msm::MSM<scalar_t, affine_t, projective_t>(rep_scalars, rep_bases, msm_size, config, &temp);
    if(res!=hipSuccess){
        std::cerr << res << "\n";
        exit(EXIT_FAILURE);
    }
    affine_t affine_temp = projective_t::to_affine(temp);
    affine_t affine_temp_mont = affine_t::ToMontgomery(affine_temp);

    Engine::G1PointAffine affine_temp2;
    std::memcpy((void*)&affine_temp2, (void*)&affine_temp_mont, sizeof(affine_temp_mont));
    G1.copy(r, affine_temp2);
}

static void bn254_icicle_cuda_g2_msm(
    Engine::G2Point &r,
    Engine::G2PointAffine *bases,
    uint8_t *scalars,
    unsigned int msm_size,
    msm::MSMConfig &config
){
    auto rep_bases = reinterpret_cast<g2_affine_t *>(bases);
    auto rep_scalars = reinterpret_cast<scalar_t *>(scalars);
    g2_projective_t temp;
    hipError_t res = msm::MSM<scalar_t, g2_affine_t, g2_projective_t>(rep_scalars, rep_bases, msm_size, config, &temp);
    if(res!=hipSuccess){
        std::cerr << res << "\n";
        exit(EXIT_FAILURE);
    }
    g2_affine_t affine_temp = g2_projective_t::to_affine(temp);
    g2_affine_t affine_temp_mont = g2_affine_t::ToMontgomery(affine_temp);
    Engine::G2PointAffine affine_temp2;
    std::memcpy((void*)&affine_temp2, (void*)&affine_temp_mont, sizeof(affine_temp_mont));
    G2.copy(r, affine_temp2);
}

template <typename Engine>
class Cuda_Prover: public Prover<Engine>{
    using Prover<Engine>::E;
    using Prover<Engine>::pointsA;
    using Prover<Engine>::nVars;
    using Prover<Engine>::pointsB1;
    using Prover<Engine>::pointsB2;
    using Prover<Engine>::pointsC;
    using Prover<Engine>::domainSize;
    using Prover<Engine>::nCoefs;
    using Prover<Engine>::coefs;
    using Prover<Engine>::fft;
    using Prover<Engine>::pointsH;
    using Prover<Engine>::vk_alpha1;
    using Prover<Engine>::vk_delta1;
    using Prover<Engine>::vk_beta2;
    using Prover<Engine>::vk_delta2;
    using Prover<Engine>::vk_beta1;
    using Prover<Engine>::nPublic;

public:  
    Cuda_Prover(Engine &_E, 
            u_int32_t _nVars, 
            u_int32_t _nPublic, 
            u_int32_t _domainSize, 
            u_int64_t _nCoefs, 
            typename Engine::G1PointAffine &_vk_alpha1,
            typename Engine::G1PointAffine &_vk_beta1,
            typename Engine::G2PointAffine &_vk_beta2,
            typename Engine::G1PointAffine &_vk_delta1,
            typename Engine::G2PointAffine &_vk_delta2,
            Coef<Engine> *_coefs, 
            typename Engine::G1PointAffine *_pointsA,
            typename Engine::G1PointAffine *_pointsB1,
            typename Engine::G2PointAffine *_pointsB2,
            typename Engine::G1PointAffine *_pointsC,
            typename Engine::G1PointAffine *_pointsH
            ): Prover<Engine>(
                _E,
                _nVars,
                _nPublic,
                _domainSize,
                _nCoefs, 
                _vk_alpha1,
                _vk_beta1,
                _vk_beta2,
                _vk_delta1,
                _vk_delta2,
                _coefs, 
                _pointsA,
                _pointsB1,
                _pointsB2,
                _pointsC,
                _pointsH

            ){

            }
    std::unique_ptr<Proof<Engine>> prove_cuda(typename Engine::FrElement *wtns);
};


template <typename Engine>
std::unique_ptr<Proof<Engine>> Cuda_Prover<Engine>::prove_cuda(typename Engine::FrElement *wtns) {

#ifdef USE_OPENMP
    START_TIMER(get_msm_config_timer);
    // msm::MSMConfig msm_config = msm::DefaultMSMConfig<affine_t>();
    device_context::DeviceContext msm_ctx = device_context::get_default_device_context();
    msm::MSMConfig msm_config = {
        msm_ctx,   // ctx
        0,     // points_size
        1,     // precompute_factor
        0,     // c
        0,     // bitsize
        10,    // large_bucket_factor
        1,     // batch_size
        false, // are_scalars_on_device
        false, // are_scalars_montgomery_form
        false, // are_points_on_device
        true, // are_points_montgomery_form
        false, // are_results_on_device
        false, // is_big_triangle
        false, // is_async
    };
    END_TIMER(get_msm_config_timer, "get MSM config");
    START_TIMER(multiexp_a_timer);
    LOG_TRACE("Start Multiexp A");
    uint32_t sW = sizeof(wtns[0]);
    typename Engine::G1Point pi_a;
    //E.g1.multiMulByScalar(pi_a, pointsA, (uint8_t *)wtns, sW, nVars);
    bn254_icicle_cuda_g1_msm(pi_a, pointsA, (uint8_t *)wtns, nVars, msm_config);
    std::ostringstream ss2;
    ss2 << "pi_a: " << E.g1.toString(pi_a);
    LOG_DEBUG(ss2);
    END_TIMER(multiexp_a_timer, "Multiexp A");

    START_TIMER(multiexp_b1_timer);
    LOG_TRACE("Start Multiexp B1");
    typename Engine::G1Point pib1;
    //E.g1.multiMulByScalar(pib1, pointsB1, (uint8_t *)wtns, sW, nVars);
    bn254_icicle_cuda_g1_msm(pib1, pointsB1,(uint8_t *)wtns, nVars, msm_config);
    std::ostringstream ss3;
    ss3 << "pib1: " << E.g1.toString(pib1);
    LOG_DEBUG(ss3);
    END_TIMER(multiexp_b1_timer, "Multiexp B1");

    START_TIMER(multiexp_b2_timer);
    LOG_TRACE("Start Multiexp B2");
    typename Engine::G2Point pi_b;
    // E.g2.multiMulByScalar(pi_b, pointsB2, (uint8_t *)wtns, sW, nVars);
    bn254_icicle_cuda_g2_msm(pi_b, pointsB2, (uint8_t *)wtns, nVars, msm_config );
    std::ostringstream ss4;
    ss4 << "pi_b: " << E.g2.toString(pi_b);
    LOG_DEBUG(ss4);
    END_TIMER(multiexp_b2_timer, "Multiexp B2");


    START_TIMER(multiexp_c_timer);
    LOG_TRACE("Start Multiexp C");
    typename Engine::G1Point pi_c;
    //E.g1.multiMulByScalar(pi_c, pointsC, (uint8_t *)((uint64_t)wtns + (nPublic +1)*sW), sW, nVars-nPublic-1);
    bn254_icicle_cuda_g1_msm(pi_c, pointsC, (uint8_t *)((uint64_t)wtns + (nPublic +1)*sW), nVars-nPublic-1, msm_config);
    std::ostringstream ss5;
    ss5 << "pi_c: " << E.g1.toString(pi_c);
    LOG_DEBUG(ss5);
    END_TIMER(multiexp_c_timer, "Multiexp C");
#else
    LOG_TRACE("Start Multiexp A");
    uint32_t sW = sizeof(wtns[0]);
    typename Engine::G1Point pi_a;
    auto pA_future = std::async([&]() {
        E.g1.multiMulByScalar(pi_a, pointsA, (uint8_t *)wtns, sW, nVars);
    });

    LOG_TRACE("Start Multiexp B1");
    typename Engine::G1Point pib1;
    auto pB1_future = std::async([&]() {
        E.g1.multiMulByScalar(pib1, pointsB1, (uint8_t *)wtns, sW, nVars);
    });

    LOG_TRACE("Start Multiexp B2");
    typename Engine::G2Point pi_b;
    auto pB2_future = std::async([&]() {
        E.g2.multiMulByScalar(pi_b, pointsB2, (uint8_t *)wtns, sW, nVars);
    });

    LOG_TRACE("Start Multiexp C");
    typename Engine::G1Point pi_c;
    auto pC_future = std::async([&]() {
        E.g1.multiMulByScalar(pi_c, pointsC, (uint8_t *)((uint64_t)wtns + (nPublic +1)*sW), sW, nVars-nPublic-1);
    });
#endif

    START_TIMER(init_a_b_c_A_timer);
    LOG_TRACE("Start Initializing a b c A");
    auto a = new typename Engine::FrElement[domainSize];
    auto b = new typename Engine::FrElement[domainSize];
    auto c = new typename Engine::FrElement[domainSize];

    #pragma omp parallel for
    for (u_int32_t i=0; i<domainSize; i++) {
        E.fr.copy(a[i], E.fr.zero());
        E.fr.copy(b[i], E.fr.zero());
    }
    END_TIMER(init_a_b_c_A_timer, "Initializing a b c A");

    START_TIMER(processing_coefs_timer);

    LOG_TRACE("Processing coefs");
#ifdef _OPENMP
    #define NLOCKS 1024
    omp_lock_t locks[NLOCKS];
    for (int i=0; i<NLOCKS; i++) omp_init_lock(&locks[i]);
    #pragma omp parallel for 
#endif
    for (u_int64_t i=0; i<nCoefs; i++) {
        typename Engine::FrElement *ab = (coefs[i].m == 0) ? a : b;
        typename Engine::FrElement aux;

        E.fr.mul(
            aux,
            wtns[coefs[i].s],
            coefs[i].coef
        );
#ifdef _OPENMP
        omp_set_lock(&locks[coefs[i].c % NLOCKS]);
#endif
        E.fr.add(
            ab[coefs[i].c],
            ab[coefs[i].c],
            aux
        );
#ifdef _OPENMP
        omp_unset_lock(&locks[coefs[i].c % NLOCKS]);
#endif
    }
#ifdef _OPENMP
    for (int i=0; i<NLOCKS; i++) omp_destroy_lock(&locks[i]);
#endif

    END_TIMER(processing_coefs_timer, "Processing coefs");

    START_TIMER(cal_c_timer);
    LOG_TRACE("Calculating c");
    #pragma omp parallel for
    for (u_int32_t i=0; i<domainSize; i++) {
        E.fr.mul(
            c[i],
            a[i],
            b[i]
        );
    }
    END_TIMER(cal_c_timer, "Calculating c");


    START_TIMER(init_fft_timer);
    LOG_TRACE("Initializing fft");
    u_int32_t domainPower = fft->log2(domainSize);
    auto ctx = device_context::get_default_device_context();
    init_icicle_cuda_ntt_ctx(ctx, domainPower);
    NTTConfig<ICICLE_S> config = DefaultNTTConfig<ICICLE_S>();
    config.ntt_algorithm = NttAlgorithm::MixedRadix;
    config.batch_size = 1;
    END_TIMER(init_fft_timer, "Initializing fft");

    START_TIMER(ifft_a_timer);
    LOG_TRACE("Start iFFT A");
    //fft->ifft(a, domainSize);
    icicle_cuda_ntt(a, domainSize,config, NTTDir::kInverse, ctx);
    END_TIMER(ifft_a_timer, "iFFT A");


    START_TIMER(a_after_ifft_timer);
    LOG_TRACE("a After ifft:");
    LOG_DEBUG(E.fr.toString(a[0]).c_str());
    LOG_DEBUG(E.fr.toString(a[1]).c_str());
    END_TIMER(a_after_ifft_timer, "a After ifft");

    START_TIMER(shift_a_timer);
    LOG_TRACE("Start Shift A");
    #pragma omp parallel for
    for (u_int64_t i=0; i<domainSize; i++) {
        E.fr.mul(a[i], a[i], fft->root(domainPower+1, i));
    }
    END_TIMER(shift_a_timer, "Shift A");

    START_TIMER(a_after_shift_timer);
    LOG_TRACE("a After shift:");
    LOG_DEBUG(E.fr.toString(a[0]).c_str());
    LOG_DEBUG(E.fr.toString(a[1]).c_str());
    END_TIMER(a_after_shift_timer, "a After shift");

    START_TIMER(fft_a_timer);
    LOG_TRACE("Start FFT A");
    //fft->fft(a, domainSize);
    icicle_cuda_ntt(a, domainSize,config, NTTDir::kForward, ctx);
    END_TIMER(fft_a_timer, "FFT A");

    START_TIMER(a_after_fft_timer);
    LOG_TRACE("a After fft:");
    LOG_DEBUG(E.fr.toString(a[0]).c_str());
    LOG_DEBUG(E.fr.toString(a[1]).c_str());
    END_TIMER(a_after_fft_timer, "a After fft");

    START_TIMER(ifft_b_timer);
    LOG_TRACE("Start iFFT B");
    //fft->ifft(b, domainSize);
    icicle_cuda_ntt(b, domainSize,config, NTTDir::kInverse, ctx);
    END_TIMER(ifft_b_timer, "iFFT B");

    START_TIMER(b_after_ifft_timer);
    LOG_TRACE("b After ifft:");
    LOG_DEBUG(E.fr.toString(b[0]).c_str());
    LOG_DEBUG(E.fr.toString(b[1]).c_str());
    END_TIMER(b_after_ifft_timer, "b After ifft");

    START_TIMER(shift_b_timer);
    LOG_TRACE("Start Shift B");
    #pragma omp parallel for
    for (u_int64_t i=0; i<domainSize; i++) {
        E.fr.mul(b[i], b[i], fft->root(domainPower+1, i));
    }
    END_TIMER(shift_b_timer, "Shift B");

    START_TIMER(b_after_shift_timer);
    LOG_TRACE("b After shift:");
    LOG_DEBUG(E.fr.toString(b[0]).c_str());
    LOG_DEBUG(E.fr.toString(b[1]).c_str());
    END_TIMER(b_after_shift_timer, "b After shift");

    START_TIMER(fft_b_timer);
    LOG_TRACE("Start FFT B");
    //fft->fft(b, domainSize);
    icicle_cuda_ntt(b, domainSize,config, NTTDir::kForward, ctx);
    END_TIMER(fft_b_timer, "FFT B");

    START_TIMER(b_after_fft_timer);
    LOG_TRACE("b After fft:");
    LOG_DEBUG(E.fr.toString(b[0]).c_str());
    LOG_DEBUG(E.fr.toString(b[1]).c_str());
    END_TIMER(b_after_fft_timer, "b After fft");

    START_TIMER(ifft_c_timer);
    LOG_TRACE("Start iFFT C");
    //fft->ifft(c, domainSize);
    icicle_cuda_ntt(c, domainSize,config, NTTDir::kInverse, ctx);
    END_TIMER(ifft_c_timer, "iFFT C");

    START_TIMER(c_after_ifft_timer);
    LOG_TRACE("c After ifft:");
    LOG_DEBUG(E.fr.toString(c[0]).c_str());
    LOG_DEBUG(E.fr.toString(c[1]).c_str());
    END_TIMER(c_after_ifft_timer, "c After ifft");

    START_TIMER(shift_c_timer);
    LOG_TRACE("Start Shift C");
    #pragma omp parallel for
    for (u_int64_t i=0; i<domainSize; i++) {
        E.fr.mul(c[i], c[i], fft->root(domainPower+1, i));
    }
    END_TIMER(shift_c_timer, "Shift C");

    START_TIMER(c_after_shift_timer);
    LOG_TRACE("c After shift:");
    LOG_DEBUG(E.fr.toString(c[0]).c_str());
    LOG_DEBUG(E.fr.toString(c[1]).c_str());
    END_TIMER(c_after_shift_timer, "c After shift");

    START_TIMER(fft_c_timer);
    LOG_TRACE("Start FFT C");
    //fft->fft(c, domainSize);
    icicle_cuda_ntt(c, domainSize,config, NTTDir::kForward, ctx);
    END_TIMER(fft_c_timer, "FFT C");

    START_TIMER(c_after_fft_timer);
    LOG_TRACE("c After fft:");
    LOG_DEBUG(E.fr.toString(c[0]).c_str());
    LOG_DEBUG(E.fr.toString(c[1]).c_str());
    END_TIMER(c_after_fft_timer, "c After fft");

    START_TIMER(start_abc_timer);
    LOG_TRACE("Start ABC");
    #pragma omp parallel for
    for (u_int64_t i=0; i<domainSize; i++) {
        E.fr.mul(a[i], a[i], b[i]);
        E.fr.sub(a[i], a[i], c[i]);
        E.fr.fromMontgomery(a[i], a[i]);
    }
    END_TIMER(start_abc_timer, "Start ABC");

    START_TIMER(abc_timer);
    LOG_TRACE("abc:");
    LOG_DEBUG(E.fr.toString(a[0]).c_str());
    LOG_DEBUG(E.fr.toString(a[1]).c_str());
    END_TIMER(abc_timer, "abc");


    delete [] b;
    delete [] c;

    START_TIMER(multiexp_h_timer);
    LOG_TRACE("Start Multiexp H");
    typename Engine::G1Point pih;
    //E.g1.multiMulByScalar(pih, pointsH, (uint8_t *)a, sizeof(a[0]), domainSize);
    bn254_icicle_cuda_g1_msm(pih, pointsH,(uint8_t *)a, domainSize, msm_config);
    std::ostringstream ss1;
    ss1 << "pih: " << E.g1.toString(pih);
    LOG_DEBUG(ss1);
    END_TIMER(multiexp_h_timer, "Multiexp H");


    delete [] a;

    typename Engine::FrElement r;
    typename Engine::FrElement s;
    typename Engine::FrElement rs;

    E.fr.copy(r, E.fr.zero());
    E.fr.copy(s, E.fr.zero());

    randombytes_buf((void *)&(r.v[0]), sizeof(r)-1);
    randombytes_buf((void *)&(s.v[0]), sizeof(s)-1);

#ifndef USE_OPENMP
    pA_future.get();
    pB1_future.get();
    pB2_future.get();
    pC_future.get();
#endif

    typename Engine::G1Point p1;
    typename Engine::G2Point p2;

    E.g1.add(pi_a, pi_a, vk_alpha1);
    E.g1.mulByScalar(p1, vk_delta1, (uint8_t *)&r, sizeof(r));
    E.g1.add(pi_a, pi_a, p1);

    E.g2.add(pi_b, pi_b, vk_beta2);
    E.g2.mulByScalar(p2, vk_delta2, (uint8_t *)&s, sizeof(s));
    E.g2.add(pi_b, pi_b, p2);

    E.g1.add(pib1, pib1, vk_beta1);
    E.g1.mulByScalar(p1, vk_delta1, (uint8_t *)&s, sizeof(s));
    E.g1.add(pib1, pib1, p1);

    E.g1.add(pi_c, pi_c, pih);

    E.g1.mulByScalar(p1, pi_a, (uint8_t *)&s, sizeof(s));
    E.g1.add(pi_c, pi_c, p1);

    E.g1.mulByScalar(p1, pib1, (uint8_t *)&r, sizeof(r));
    E.g1.add(pi_c, pi_c, p1);

    E.fr.mul(rs, r, s);
    E.fr.toMontgomery(rs, rs);

    E.g1.mulByScalar(p1, vk_delta1, (uint8_t *)&rs, sizeof(rs));
    E.g1.sub(pi_c, pi_c, p1);

    Proof<Engine> *p = new Proof<Engine>(Engine::engine);
    E.g1.copy(p->A, pi_a);
    E.g2.copy(p->B, pi_b);
    E.g1.copy(p->C, pi_c);

    return std::unique_ptr<Proof<Engine>>(p);    

}


template <typename Engine>
std::unique_ptr<Cuda_Prover<Engine>> makeCuda_Prover(
    u_int32_t nVars, 
    u_int32_t nPublic, 
    u_int32_t domainSize, 
    u_int64_t nCoeffs, 
    void *vk_alpha1,
    void *vk_beta_1,
    void *vk_beta_2,
    void *vk_delta_1,
    void *vk_delta_2,
    void *coefs, 
    void *pointsA, 
    void *pointsB1, 
    void *pointsB2, 
    void *pointsC, 
    void *pointsH
) {
    Cuda_Prover<Engine> *p = new Cuda_Prover<Engine>(
        Engine::engine, 
        nVars, 
        nPublic, 
        domainSize, 
        nCoeffs, 
        *(typename Engine::G1PointAffine *)vk_alpha1,
        *(typename Engine::G1PointAffine *)vk_beta_1,
        *(typename Engine::G2PointAffine *)vk_beta_2,
        *(typename Engine::G1PointAffine *)vk_delta_1,
        *(typename Engine::G2PointAffine *)vk_delta_2,
        (Coef<Engine> *)((uint64_t)coefs + 4), 
        (typename Engine::G1PointAffine *)pointsA,
        (typename Engine::G1PointAffine *)pointsB1,
        (typename Engine::G2PointAffine *)pointsB2,
        (typename Engine::G1PointAffine *)pointsC,
        (typename Engine::G1PointAffine *)pointsH
    );
    return std::unique_ptr< Cuda_Prover<Engine> >(p);
}

#endif